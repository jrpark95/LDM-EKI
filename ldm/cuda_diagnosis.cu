#include <hip/hip_runtime.h>
#include <iostream>

void printCudaError(hipError_t error, const char* operation) {
    std::cout << operation << " failed with error: " << hipGetErrorString(error) 
              << " (code " << error << ")" << std::endl;
}

int main() {
    std::cout << "=== CUDA Diagnosis ===" << std::endl;
    
    // 1. Get device count
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printCudaError(err, "hipGetDeviceCount");
        return 1;
    }
    std::cout << "Device count: " << deviceCount << std::endl;
    
    // 2. Try to get current device
    int currentDevice;
    err = hipGetDevice(&currentDevice);
    if (err != hipSuccess) {
        printCudaError(err, "hipGetDevice");
    } else {
        std::cout << "Current device: " << currentDevice << std::endl;
    }
    
    // 3. Try to set device 0
    err = hipSetDevice(0);
    if (err != hipSuccess) {
        printCudaError(err, "cudaSetDevice(0)");
        
        // Try other devices
        for (int i = 1; i < deviceCount && i < 8; i++) {
            err = hipSetDevice(i);
            if (err == hipSuccess) {
                std::cout << "Successfully set device " << i << std::endl;
                break;
            } else {
                printCudaError(err, ("cudaSetDevice(" + std::to_string(i) + ")").c_str());
            }
        }
        
        if (err != hipSuccess) {
            std::cout << "Failed to set any device" << std::endl;
            return 1;
        }
    }
    
    // 4. Get device properties
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess) {
        printCudaError(err, "hipGetDeviceProperties");
    } else {
        std::cout << "Device 0: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "Total memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
    }
    
    // 5. Try minimal memory allocation
    void* d_ptr = nullptr;
    err = hipMalloc(&d_ptr, 4);
    if (err != hipSuccess) {
        printCudaError(err, "cudaMalloc(4 bytes)");
        return 1;
    }
    std::cout << "Successfully allocated 4 bytes at " << d_ptr << std::endl;
    
    // 6. Try to free
    err = hipFree(d_ptr);
    if (err != hipSuccess) {
        printCudaError(err, "hipFree");
        return 1;
    }
    std::cout << "Successfully freed memory" << std::endl;
    
    std::cout << "=== CUDA Test PASSED ===" << std::endl;
    return 0;
}
