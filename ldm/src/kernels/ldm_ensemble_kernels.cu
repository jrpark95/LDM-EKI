#include "hip/hip_runtime.h"
#include "ldm.cuh"
#include "ldm_ensemble_init.cuh"

// Ensemble activation kernel implementation
__global__ void update_particle_flags_ensembles(LDM::LDMpart* d_part,
                                               int nop_per_ensemble,
                                               int Nens,
                                               float activationRatio) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = nop_per_ensemble * Nens;
    
    if (idx >= total) return;
    
    int local_i = idx % nop_per_ensemble;
    int maxLocal = static_cast<int>(nop_per_ensemble * activationRatio);
    
    d_part[idx].flag = (local_i < maxLocal) ? 1 : 0;
}

// Sanity check kernel implementation
__global__ void count_active_particles_per_ensemble(const LDM::LDMpart* d_part,
                                                   int nop_per_ensemble,
                                                   int Nens,
                                                   int* active_counts) {
    int ensemble_idx = blockIdx.x;
    if (ensemble_idx >= Nens) return;
    
    int tid = threadIdx.x;
    int local_count = 0;
    
    // Count active particles in this ensemble
    for (int i = tid; i < nop_per_ensemble; i += blockDim.x) {
        int global_idx = ensemble_idx * nop_per_ensemble + i;
        if (d_part[global_idx].flag == 1) {
            local_count++;
        }
    }
    
    // Reduce within block
    __shared__ int sdata[256];
    sdata[tid] = local_count;
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        active_counts[ensemble_idx] = sdata[0];
    }
}